#include "belief-propagation.hpp"

static uint64_t getHostHash(const char* string) {
    // Based on DJB2, result = result * 33 + char
    uint64_t result = 5381;
    for (int c = 0; string[c] != '\0'; c++){
        result = ((result << 5) + result) + string[c];
    }
    return result;
}

static void getHostName(char* hostname, int maxlen) {
    gethostname(hostname, maxlen);
    for (int i=0; i< maxlen; i++) {
        if (hostname[i] == '.') {
            hostname[i] = '\0';
            return;
        }
    }
}

int main(int argc, char *argv[]) {
    int my_rank, n_ranks, local_rank, num_devices = 0;
    struct hipDeviceProp_t prop;

    // initialize MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &my_rank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &n_ranks));

    CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, 0));

    // calculate local rank
    uint64_t host_hashes[n_ranks];
    char hostname[1024];
    getHostName(hostname, 1024);
    host_hashes[my_rank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, host_hashes, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for(int p = 0; p < n_ranks; p++) {
        if (p == my_rank) {
            break;
        }
        if (host_hashes[p] == host_hashes[my_rank]) {
            local_rank++;
        }
    }

    // get the number of devices
    CUDA_CHECK_RETURN(hipGetDeviceCount(&num_devices));

    // start the fun
    FILE * out = fopen("cuda_benchmark_loopy_edge_openmpi.csv", "w");
    fprintf(out, "File Name,Propagation Type,Number of Nodes,Number of Edges,Diameter,Number of Iterations,BP Run Time(s)\n");
    fflush(out);

    run_test_loopy_belief_propagagtion_mtx_file_edge_openmpi_cuda("/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/10_20.bif.edges.mtx", "/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/10_20.bif.nodes.mtx", out, my_rank, n_ranks, num_devices);
    run_test_loopy_belief_propagagtion_mtx_file_edge_openmpi_cuda("/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/100_200.bif.edges.mtx", "/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/100_200.bif.nodes.mtx", out, my_rank, n_ranks, num_devices);
    run_test_loopy_belief_propagagtion_mtx_file_edge_openmpi_cuda("/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/1000_2000.bif.edges.mtx", "/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/1000_2000.bif.nodes.mtx", out, my_rank, n_ranks, num_devices);
    run_test_loopy_belief_propagagtion_mtx_file_edge_openmpi_cuda("/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/10000_20000.bif.edges.mtx", "/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/10000_20000.bif.nodes.mtx", out, my_rank, n_ranks, num_devices);
    run_test_loopy_belief_propagagtion_mtx_file_edge_openmpi_cuda("/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/100000_200000.bif.edges.mtx", "/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/100000_200000.bif.nodes.mtx", out, my_rank, n_ranks, num_devices);
    run_test_loopy_belief_propagagtion_mtx_file_edge_openmpi_cuda("/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/200000_400000.bif.edges.mtx", "/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/200000_400000.bif.nodes.mtx", out, my_rank, n_ranks, num_devices);
    run_test_loopy_belief_propagagtion_mtx_file_edge_openmpi_cuda("/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/400000_800000.bif.edges.mtx", "/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/400000_800000.bif.nodes.mtx", out, my_rank, n_ranks, num_devices);
    run_test_loopy_belief_propagagtion_mtx_file_edge_openmpi_cuda("/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/800000_1600000.bif.edges.mtx", "/mnt/raid0_huge/micheal/gunrock_benchmark_files_copy/800000_1600000.bif.nodes.mtx", out, my_rank, n_ranks, num_devices);
    
    MPI_Finalize();
    return EXIT_SUCCESS;
}