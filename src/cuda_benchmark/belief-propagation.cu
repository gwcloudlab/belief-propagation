#include "hip/hip_runtime.h"
#include "belief-propagation.hpp"

__device__ __forceinline__ unsigned int LaneMaskLt()
{
    unsigned int ret;
    asm ("mov.u32 %0, %%lanemask_lt;" : "=r"(ret) );
    return ret;
}



__device__
unsigned int atomic_add_inc(unsigned int * ctr) {
    // from https://devblogs.nvidia.com/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
    unsigned int active = __activemask();
    int leader = __ffs(active) - 1;
    int change = __popc(active);
    unsigned int rank = __popc(active & LaneMaskLt());
    unsigned int warp_res;
    if(rank == 0) {
        warp_res = atomicAdd(ctr, change);
    }
    warp_res = __shfl_sync(active, warp_res, leader);
    return warp_res + rank;
}


__device__
void update_work_queue_nodes_cuda(unsigned int * work_queue_nodes, unsigned int * num_work_items, unsigned int *work_queue_scratch, struct belief * node_states, unsigned int num_vertices, float precision, unsigned int *ctr) {
    unsigned int i;
    unsigned int index_ctr;
    unsigned int orig_num_work_items = *num_work_items;

    for(i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_work_items; i += blockDim.x * gridDim.x){
        if(fabs(node_states[work_queue_nodes[i]].current - node_states[work_queue_nodes[i]].previous) >= precision) {
            index_ctr = atomic_add_inc(ctr);
            work_queue_scratch[index_ctr] = work_queue_nodes[i];
        }
    }

    __syncthreads();
    for(i = blockIdx.x * blockDim.x + threadIdx.x; i < num_vertices; i += blockDim.x * gridDim.x){
        work_queue_nodes[i] = work_queue_scratch[i];
    }
    atomicCAS(num_work_items, orig_num_work_items, *ctr);
}

__device__
void update_work_queue_edges_cuda(unsigned int * work_queue_edge, unsigned int * num_work_items, unsigned int *work_queue_scratch, struct belief * edge_states, unsigned int num_edges, float precision, unsigned int *ctr) {
    unsigned int i;
    unsigned int index_ctr;
    unsigned int orig_num_work_items = *num_work_items;

    for(i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_work_items; i += blockDim.x * gridDim.x){
        if(fabs(edge_states[work_queue_edge[i]].current - edge_states[work_queue_edge[i]].previous) >= precision) {
            index_ctr = atomic_add_inc(ctr);
            work_queue_scratch[index_ctr] = work_queue_edge[i];
        }
    }

    __syncthreads();
    for(i = blockIdx.x * blockDim.x + threadIdx.x; i < num_edges; i += blockDim.x * gridDim.x){
        work_queue_edge[i] = work_queue_scratch[i];
    }
    atomicCAS(num_work_items, orig_num_work_items, *ctr);
}


/**
 * Initialize the message buffer to what is stored in node_states
 * @param buffer The message buffer
 * @param node_states The states to init to
 * @param num_variables The size of the arrays
 * @param node_index The index of the current belief
  */
__device__
void init_message_buffer_cuda(struct belief *buffer, struct belief *node_states, unsigned int num_variables, unsigned int node_index){
    unsigned int j;

    buffer->size = num_variables;
    for(j = 0; j < num_variables; ++j){
        buffer->data[j] = node_states[node_index].data[j];
    }

}

/**
 * Combine the dest node with the incoming belief
 * @param dest The belief to update
 * @param edge_messages The incoming belief
 * @param length The number of probabilities to combine
 * @param offset The offset in the incoming messages
 */
__device__
void combine_message_cuda(struct belief * dest, struct belief * edge_messages, unsigned int length, unsigned int offset){
    unsigned int i;
    float message;
    __shared__ float buffer[BLOCK_SIZE];

    for(i = 0; i < length; ++i){
        buffer[threadIdx.x] = dest->data[i];
        message = edge_messages[offset].data[i];
        if(message == message){
            buffer[threadIdx.x] *= message;
            dest->data[i] = buffer[threadIdx.x];
        }
    }
}

__device__
void combine_page_rank_message_cuda(struct belief * dest, struct belief * edge_messages, unsigned int length, unsigned int offset){
    unsigned int i;
    float message;
    __shared__ float buffer[BLOCK_SIZE];

    for(i = 0; i < length; ++i){
        buffer[threadIdx.x] = dest->data[i];
        message = edge_messages[offset].data[i];
        if(message == message){
            buffer[threadIdx.x] += message;
            dest->data[i] = buffer[threadIdx.x];
        }
    }
}

__device__
void combine_viterbi_message_cuda(struct belief * dest, struct belief * edge_messages, unsigned int length, unsigned int offset){
    unsigned int i;
    float message;
    __shared__ float buffer[BLOCK_SIZE];

    for(i = 0; i < length; ++i){
        buffer[threadIdx.x] = dest->data[i];
        message = edge_messages[offset].data[i];
        if(message == message){
            buffer[threadIdx.x] = fmaxf(buffer[threadIdx.x], message);
            dest->data[i] = buffer[threadIdx.x];
        }
    }
}

/**
 * Combines the incoming messages for the given node
 * @param message_buffer The current belief
 * @param dest_nodes_to_edges_nodes The indices in dest_nodes_to_edges_edges by node index
 * @param dest_nodes_to_edges_edges The indices of the edges indexed by their dest node
 * @param current_num_edges The number of edges in the graph
 * @param num_vertices The number of vertices in the graph
 * @param num_variables The number of beliefs in the graph
 * @param idx The index of the current node
 */
__device__
void read_incoming_messages_cuda(struct belief * message_buffer,
                                 struct belief * previous_messages,
                                 unsigned int * dest_nodes_to_edges_nodes,
                                 unsigned int * dest_nodes_to_edges_edges,
                                 unsigned int current_num_edges,
                            unsigned int num_vertices, unsigned int num_variables, unsigned int idx){
    unsigned int start_index, end_index, j, edge_index;

    start_index = dest_nodes_to_edges_nodes[idx];
    if(idx + 1 >= num_vertices){
        end_index = current_num_edges;
    }
    else{
        end_index = dest_nodes_to_edges_nodes[idx + 1];
    }
    for(j = start_index; j < end_index; ++j){
        edge_index = dest_nodes_to_edges_edges[j];

        combine_message_cuda(message_buffer, previous_messages, num_variables, edge_index);
    }
}

/**
 * Send the current beliefs along the edge to the current node
 * @param buffer The current node
 * @param edge_index The index of the edge
 * @param joint_probabilities The joint probability table on the edge
 * @param edge_messages The current beliefs
 */
__device__
void send_message_for_edge_cuda(struct belief * buffer, unsigned int edge_index,
                                struct joint_probability * joint_probabilities,
                                struct belief * edge_messages){
    unsigned int i, j, num_src, num_dest;
    float sum;
    struct joint_probability joint_probability;
    __shared__ float partial_sums[BLOCK_SIZE * MAX_STATES];

    joint_probability = joint_probabilities[edge_index];

    num_src = joint_probability.dim_x;
    num_dest = joint_probability.dim_y;

    sum = 0.0;
    for(i = 0; i < num_src; ++i){
        partial_sums[threadIdx.x * MAX_STATES + i] = 0.0;
        for(j = 0; j < num_dest; ++j){
            partial_sums[threadIdx.x * MAX_STATES + i] += joint_probability.data[i][j] * buffer->data[j];
        }
        sum += partial_sums[threadIdx.x * MAX_STATES + i];
    }
    if(sum <= 0.0){
        sum = 1.0;
    }
    edge_messages[edge_index].previous = edge_messages[edge_index].current;
    edge_messages[edge_index].current = sum;
    for(i = 0; i < num_src; ++i){
        edge_messages[edge_index].data[i] /= sum;
    }
}

/**
 * Propagate the current beliefs to current node
 * @param message_buffer The current node
 * @param current_num_edges The number of edges in the graph
 * @param joint_probabilities The list of joint probabilities
 * @param current_edge_messages The incoming messages
 * @param src_nodes_to_edges_nodes The indices in src_nodes_to_edges_edges indexed by src node index
 * @param src_nodes_to_edges_edges The edges indexed by their source node
 * @param num_vertices The number of the vertices in the graph
 * @param idx The current node index
 */
__device__
void send_message_for_node_cuda(struct belief *message_buffer, unsigned int current_num_edges,
                                struct joint_probability *joint_probabilities,
                                struct belief *current_edge_messages,
                                unsigned int * src_nodes_to_edges_nodes, unsigned int * src_nodes_to_edges_edges,
                                unsigned int num_vertices, unsigned int idx){
    unsigned int start_index, end_index, j, edge_index;

    start_index = src_nodes_to_edges_nodes[idx];
    if(idx + 1 >= num_vertices){
        end_index = current_num_edges;
    }
    else{
        end_index = src_nodes_to_edges_nodes[idx + 1];
    }

    for(j = start_index; j < end_index; ++j){
        edge_index = src_nodes_to_edges_edges[j];
        send_message_for_edge_cuda(message_buffer, edge_index, joint_probabilities, current_edge_messages);
    }
}

/**
 * Marginalizes and normalizes the belief probabilities for a given node
 * @param node_num_vars The number of variables for a given node
 * @param node_states The states of the given node
 * @param idx The node's index
 * @param current_edges_messages The array holding the current beliefs on the ege
 * @param dest_nodes_to_edges_nodes The parallel array holding the mapping of nodes to their edges in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges Array holding the mapping of nodes to their edges in which they are the destination
 * @param num_vertices The number of vertices (nodes) in the graph
 * @param num_edges The number of edges in the graph
 */
__device__
void marginalize_node(struct belief *node_states, unsigned int idx,
                      struct belief *current_edges_messages,
                      unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                      unsigned int num_vertices, unsigned int num_edges){
    unsigned int i, num_variables, start_index, end_index, edge_index;
    float sum;

    num_variables = node_states[idx].size;

    struct belief new_belief;

    new_belief.size = num_variables;
    for(i = 0; i < num_variables; ++i){
        new_belief.data[i] = 1.0;
    }

    start_index = dest_nodes_to_edges_nodes[idx];
    if(idx + 1 >= num_vertices){
        end_index = num_edges;
    }
    else{
        end_index = dest_nodes_to_edges_nodes[idx + 1];
    }

    for(i = start_index; i < end_index; ++i){
        edge_index = dest_nodes_to_edges_edges[i];

        combine_message_cuda(&new_belief, current_edges_messages, num_variables, edge_index);
    }
    if(start_index < end_index){
        for(i = 0; i < num_variables; ++i){
             new_belief.data[i] *= node_states[idx].data[i];
        }
    }
    sum = 0.0;
    for(i = 0; i < num_variables; ++i){
        sum += new_belief.data[i];
    }
    if(sum <= 0.0){
        sum = 1.0;
    }

    node_states[idx].previous = node_states[idx].current;
    node_states[idx].current = sum;
    for(i = 0; i < num_variables; ++i){
        node_states[idx].data[i] /= sum;
    }
}

/**
 * Marginalizes and normalizes the PageRanks for a given node
 * @param node_num_vars The number of variables for a given node
 * @param node_states The states of the given node
 * @param idx The node's index
 * @param current_edges_messages The array holding the current beliefs on the ege
 * @param dest_nodes_to_edges_nodes The parallel array holding the mapping of nodes to their edges in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges Array holding the mapping of nodes to their edges in which they are the destination
 * @param num_vertices The number of vertices (nodes) in the graph
 * @param num_edges The number of edges in the graph
 */
__device__
void marginalize_page_rank_node(struct belief *node_states, unsigned int idx,
                                struct belief *current_edges_messages,
                                unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                                unsigned int num_vertices, unsigned int num_edges) {
    unsigned int i, num_variables, start_index, end_index, edge_index;
    float factor;

    num_variables = node_states[idx].size;

    struct belief new_belief;

    new_belief.size = num_variables;
    for (i = 0; i < num_variables; ++i) {
        new_belief.data[i] = 0.0;
    }

    start_index = dest_nodes_to_edges_nodes[idx];
    if (idx + 1 >= num_vertices) {
        end_index = num_edges;
    } else {
        end_index = dest_nodes_to_edges_nodes[idx + 1];
    }

    for (i = start_index; i < end_index; ++i) {
        edge_index = dest_nodes_to_edges_edges[i];

        combine_page_rank_message_cuda(&new_belief, current_edges_messages, num_variables, edge_index);
    }

    if (start_index < end_index) {
        factor = (1 - DAMPENING_FACTOR) / (end_index - start_index);
        for (i = 0; i < num_variables; ++i) {
            new_belief.data[i] = factor + DAMPENING_FACTOR * new_belief.data[i];
        }
    }
}

/**
 * Computes the argmax the belief probabilities for a given node
 * @param node_num_vars The number of variables for a given node
 * @param node_states The states of the given node
 * @param idx The node's index
 * @param current_edges_messages The array holding the current beliefs on the ege
 * @param dest_nodes_to_edges_nodes The parallel array holding the mapping of nodes to their edges in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges Array holding the mapping of nodes to their edges in which they are the destination
 * @param num_vertices The number of vertices (nodes) in the graph
 * @param num_edges The number of edges in the graph
 */
__device__
void argmax_node(struct belief *node_states, unsigned int idx,
                      struct belief *current_edges_messages,
                      unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                      unsigned int num_vertices, unsigned int num_edges){
    unsigned int i, num_variables, start_index, end_index, edge_index;

    num_variables = node_states[idx].size;

    struct belief new_belief;

    new_belief.size = num_variables;
    for(i = 0; i < num_variables; ++i){
        new_belief.data[i] = -1.0f;
    }

    start_index = dest_nodes_to_edges_nodes[idx];
    if(idx + 1 >= num_vertices){
        end_index = num_edges;
    }
    else{
        end_index = dest_nodes_to_edges_nodes[idx + 1];
    }

    for(i = start_index; i < end_index; ++i){
        edge_index = dest_nodes_to_edges_edges[i];

        combine_viterbi_message_cuda(&new_belief, current_edges_messages, num_variables, edge_index);
    }
    if(start_index < end_index){
        for(i = 0; i < num_variables; ++i){
            new_belief.data[i] = fmaxf(new_belief.data[i], node_states[idx].data[i]);
        }
    }
}

/**
 * Marginalizes and normalizes all nodes in the graph
 * @param node_states The current states of all nodes in the graph
 * @param current_edges_messages The current messages held in transit along the edge
 * @param dest_nodes_to_edges_nodes The mapping of nodes to their edges; parallel array which maps nodes to their edge indices in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges The mapping nodes of nodes to the edges; consists of edge indices for which the node is the destination node
 * @param num_vertices The number of vertices in the graph
 * @param num_edges The number of edges in the graph
 */
__global__
void marginalize_nodes(struct belief *node_states,
                       struct belief *current_edges_messages,
                       unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                       unsigned int num_vertices, unsigned int num_edges) {
    unsigned int idx;
    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_vertices; idx += blockDim.x * gridDim.x){
        marginalize_node(node_states, idx, current_edges_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
    }
}

/**
 * Marginalizes and normalizes all PageRank nodes in the graph
 * @param node_states The current states of all nodes in the graph
 * @param current_edges_messages The current messages held in transit along the edge
 * @param dest_nodes_to_edges_nodes The mapping of nodes to their edges; parallel array which maps nodes to their edge indices in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges The mapping nodes of nodes to the edges; consists of edge indices for which the node is the destination node
 * @param num_vertices The number of vertices in the graph
 * @param num_edges The number of edges in the graph
 */
__global__
void marginalize_page_rank_nodes(struct belief *node_states,
                       struct belief *current_edges_messages,
                       unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                       unsigned int num_vertices, unsigned int num_edges) {
    unsigned int idx;
    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_vertices; idx += blockDim.x * gridDim.x){
        marginalize_page_rank_node(node_states, idx, current_edges_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
    }
}

/**
 * Computes the argmax all nodes in the graph
 * @param node_states The current states of all nodes in the graph
 * @param current_edges_messages The current messages held in transit along the edge
 * @param dest_nodes_to_edges_nodes The mapping of nodes to their edges; parallel array which maps nodes to their edge indices in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges The mapping nodes of nodes to the edges; consists of edge indices for which the node is the destination node
 * @param num_vertices The number of vertices in the graph
 * @param num_edges The number of edges in the graph
 */
__global__
void argmax_nodes(struct belief *node_states,
                       struct belief *current_edges_messages,
                       unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                       unsigned int num_vertices, unsigned int num_edges) {
    unsigned int idx;
    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_vertices; idx += blockDim.x * gridDim.x){
        argmax_node(node_states, idx, current_edges_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
    }
}

/**
 * Runs loopy BP on the GPU
 * @param num_vertices The number of vertices (nodes) in the graph
 * @param num_edges The number of edges in the graph
 * @param node_messages The current beliefs of each node
 * @param joint_probabilities The joint probability table for each edge
 * @param previous_edge_messages The previous messages sent on the edges
 * @param current_edge_messages The current messages sent on the edges
 * @param src_nodes_to_edges_nodes The mapping of source nodes to their edges; parallel array; mapping of nodes to their edges in src_nodes_to_edges_edges
 * @param src_nodes_to_edges_edges The mapping of source nodes to their edges; consists of edges indexed by their source nodes
 * @param dest_nodes_to_edges_nodes The mapping of dest nodes to their edges; parallel array; mapping of nodes to their edges in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges THe mapping of dest nodes to their edges; consists of edges indexed by their dest nodes
 */
__global__
void loopy_propagate_main_loop(unsigned int num_vertices, unsigned int num_edges,
                               struct belief *node_messages,
                               struct joint_probability *joint_probabilities,
                               struct belief *current_edge_messages,
                               unsigned int *work_queue_nodes, unsigned int *num_work_items,
                               unsigned int *work_queue_scratch,
                               unsigned int * src_nodes_to_edges_nodes, unsigned int * src_nodes_to_edges_edges,
                               unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                               unsigned int * ctr){
    unsigned int i, idx, num_variables;
    struct belief new_belief;

    for(i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_work_items; i += blockDim.x * gridDim.x){
        idx = work_queue_nodes[i];

        num_variables = node_messages[idx].size;

        init_message_buffer_cuda(&new_belief, node_messages, num_variables, idx);
        __syncthreads();

        read_incoming_messages_cuda(&new_belief, current_edge_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_edges, num_vertices, num_variables, idx);
        __syncthreads();

        send_message_for_node_cuda(&new_belief, num_edges, joint_probabilities, current_edge_messages, src_nodes_to_edges_nodes, src_nodes_to_edges_edges, num_vertices, idx);
        __syncthreads();

        marginalize_node(node_messages, idx, current_edge_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
        __syncthreads();
    }


    update_work_queue_nodes_cuda(work_queue_nodes, num_work_items, work_queue_scratch, node_messages, num_vertices, PRECISION_ITERATION, ctr);

    __syncthreads();


}

__global__
void loopy_propagate_partitioned_read_main_loop(unsigned int num_vertices, unsigned int num_edges,
                                                struct belief *node_messages,
                                                struct belief *current_edge_messages,
                                                unsigned int *work_queue_nodes, unsigned int *num_work_items,
                                                unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                                                struct belief *node_buffer,
                                                idx_t * nodes_to_partitions, idx_t current_partition) {
    unsigned int i, idx, num_variables;


    for (i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_work_items; i += blockDim.x * gridDim.x) {
        idx = work_queue_nodes[i];
        if (nodes_to_partitions[idx] == current_partition) {

            num_variables = node_messages[idx].size;

            init_message_buffer_cuda(&node_buffer[idx], node_messages, num_variables, idx);
            __syncthreads();

            read_incoming_messages_cuda(&node_buffer[idx], current_edge_messages, dest_nodes_to_edges_nodes,
                                        dest_nodes_to_edges_edges, num_edges, num_vertices, num_variables, idx);
            __syncthreads();
        }
    }

    __syncthreads();
}

__global__
void loopy_propagate_partitioned_send_main_loop(unsigned int num_vertices, unsigned int num_edges,
                                                struct belief *node_messages,
                                                struct joint_probability *joint_probabilities,
                                                struct belief *current_edge_messages,
                                                unsigned int *work_queue_nodes, unsigned int *num_work_items,
                                                unsigned int *work_queue_scratch,
                                                unsigned int * src_nodes_to_edges_nodes, unsigned int * src_nodes_to_edges_edges,
                                                unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges,
                                                struct belief *node_buffer,
                                                idx_t * nodes_to_partitions, idx_t current_partition){
    unsigned int i, idx, num_variables;
    for(i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_work_items; i += blockDim.x * gridDim.x) {
        idx = work_queue_nodes[i];

        if(nodes_to_partitions[idx] == current_partition) {
            send_message_for_node_cuda(&node_buffer[idx], num_edges, joint_probabilities, current_edge_messages, src_nodes_to_edges_nodes, src_nodes_to_edges_edges, num_vertices, idx);
            __syncthreads();

            marginalize_node(node_messages, idx, current_edge_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
            __syncthreads();
        }
    }

    __syncthreads();
}

__global__
void loopy_propagate_partitioned_update(unsigned int num_vertices,
                                        struct belief *node_messages,
                                        unsigned int *work_queue_nodes, unsigned int *num_work_items,
                                        unsigned int *work_queue_scratch, unsigned int *ctr) {
    update_work_queue_nodes_cuda(work_queue_nodes, num_work_items, work_queue_scratch, node_messages, num_vertices, PRECISION_ITERATION, ctr);
    __syncthreads();
}


/**
 * Runs PageRank on the GPU
 * @param num_vertices The number of vertices (nodes) in the graph
 * @param num_edges The number of edges in the graph
 * @param node_messages The current beliefs of each node
 * @param joint_probabilities The joint probability table for each edge
 * @param previous_edge_messages The previous messages sent on the edges
 * @param current_edge_messages The current messages sent on the edges
 * @param src_nodes_to_edges_nodes The mapping of source nodes to their edges; parallel array; mapping of nodes to their edges in src_nodes_to_edges_edges
 * @param src_nodes_to_edges_edges The mapping of source nodes to their edges; consists of edges indexed by their source nodes
 * @param dest_nodes_to_edges_nodes The mapping of dest nodes to their edges; parallel array; mapping of nodes to their edges in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges THe mapping of dest nodes to their edges; consists of edges indexed by their dest nodes
 */
__global__
void page_rank_main_loop(unsigned int num_vertices, unsigned int num_edges,
                               struct belief *node_messages,
                               struct joint_probability *joint_probabilities,
                               struct belief *current_edge_messages,
                               unsigned int * src_nodes_to_edges_nodes, unsigned int * src_nodes_to_edges_edges,
                               unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges){
    unsigned int idx, num_variables;
    struct belief new_belief;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_vertices; idx += blockDim.x * gridDim.x){
        num_variables = node_messages[idx].size;

        init_message_buffer_cuda(&new_belief, node_messages, num_variables, idx);
        __syncthreads();

        read_incoming_messages_cuda(&new_belief, current_edge_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_edges, num_vertices, num_variables, idx);
        __syncthreads();

        send_message_for_node_cuda(&new_belief, num_edges, joint_probabilities, current_edge_messages, src_nodes_to_edges_nodes, src_nodes_to_edges_edges, num_vertices, idx);
        __syncthreads();

        marginalize_page_rank_node(node_messages, idx, current_edge_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
        __syncthreads();
    }
}

/**
 * Runs Viterbi on the GPU
 * @param num_vertices The number of vertices (nodes) in the graph
 * @param num_edges The number of edges in the graph
 * @param node_messages The current beliefs of each node
 * @param joint_probabilities The joint probability table for each edge
 * @param previous_edge_messages The previous messages sent on the edges
 * @param current_edge_messages The current messages sent on the edges
 * @param src_nodes_to_edges_nodes The mapping of source nodes to their edges; parallel array; mapping of nodes to their edges in src_nodes_to_edges_edges
 * @param src_nodes_to_edges_edges The mapping of source nodes to their edges; consists of edges indexed by their source nodes
 * @param dest_nodes_to_edges_nodes The mapping of dest nodes to their edges; parallel array; mapping of nodes to their edges in dest_nodes_to_edges_edges
 * @param dest_nodes_to_edges_edges THe mapping of dest nodes to their edges; consists of edges indexed by their dest nodes
 */
__global__
void viterbi_main_loop(unsigned int num_vertices, unsigned int num_edges,
                         struct belief *node_messages,
                         struct joint_probability *joint_probabilities,
                         struct belief *current_edge_messages,
                         unsigned int * src_nodes_to_edges_nodes, unsigned int * src_nodes_to_edges_edges,
                         unsigned int * dest_nodes_to_edges_nodes, unsigned int * dest_nodes_to_edges_edges){
    unsigned int idx, num_variables;
    struct belief new_belief;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_vertices; idx += blockDim.x * gridDim.x){
        num_variables = node_messages[idx].size;

        init_message_buffer_cuda(&new_belief, node_messages, num_variables, idx);
        __syncthreads();

        read_incoming_messages_cuda(&new_belief, current_edge_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_edges, num_vertices, num_variables, idx);
        __syncthreads();

        send_message_for_node_cuda(&new_belief, num_edges, joint_probabilities, current_edge_messages, src_nodes_to_edges_nodes, src_nodes_to_edges_edges, num_vertices, idx);
        __syncthreads();

        argmax_node(node_messages, idx, current_edge_messages, dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
        __syncthreads();
    }
}

/**
 * Updates the current belief using the joint probability and the incoming messages
 * @param belief The belief being updated
 * @param src_index The index of the incoming belief message
 * @param edge_index The index of the edge carrying the belief
 * @param joint_probabilities The joint probability matrix
 * @param edge_messages The current message on the edge
 */
__device__
static void send_message_for_edge_iteration_cuda(struct belief *belief, unsigned int src_index, unsigned int edge_index,
                                                 struct joint_probability *joint_probabilities, struct belief *edge_messages){
    unsigned int i, j, num_src, num_dest;
    float sum;
    __shared__ float partial_sums[MAX_STATES * BLOCK_SIZE];

    num_src = joint_probabilities[edge_index].dim_x;
    num_dest = joint_probabilities[edge_index].dim_y;

    sum = 0.0;
    for(i = 0; i < num_src; ++i){
        partial_sums[MAX_STATES * threadIdx.x + i] = 0.0;
        for(j = 0; j < num_dest; ++j){
            partial_sums[MAX_STATES * threadIdx.x + i] += joint_probabilities[edge_index].data[i][j] * belief[src_index].data[j];
        }
        sum += partial_sums[MAX_STATES * threadIdx.x + i];
    }
    if(sum <= 0.0){
        sum = 1.0;
    }
    edge_messages[edge_index].previous = edge_messages[edge_index].current;
    edge_messages[edge_index].current = sum;
    for (i = 0; i < num_src; ++i) {
        edge_messages[edge_index].data[i] /= sum;
    }
}

/**
 * Sends a message along the edge
 * @param num_edges The number of edges in the graph
 * @param edges_src_index The index of the source node for the edge
 * @param node_states The beliefs of all nodes in the graph
 * @param joint_probabilities The joint probabilities of all edges in the graph
 * @param current_edge_messages The current belief held on the edge
 */
__global__
void send_message_for_edge_iteration_cuda_kernel(unsigned int num_edges, unsigned int * edges_src_index,
                                                 struct belief *node_states,
                                                 struct joint_probability *joint_probabilities,
                                                 struct belief *current_edge_messages){
    unsigned int idx, src_node_index;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_edges; idx += blockDim.x * gridDim.x){
        src_node_index = edges_src_index[idx];

        send_message_for_edge_iteration_cuda(node_states, src_node_index, idx, joint_probabilities, current_edge_messages);
    }
}

__global__
void send_message_for_edge_iteration_cuda_work_queue_kernel(unsigned int num_edges, unsigned int * edges_src_index,
                                                            struct belief *node_states,
                                                            struct joint_probability *joint_probabilities,
                                                            struct belief *current_edge_messages,
                                                            unsigned int * work_queue_edges, unsigned int * num_work_queue_items) {
    unsigned int i, idx, src_node_index;
    for(i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_work_queue_items; i += blockDim.x * gridDim.x){
        idx = work_queue_edges[i];

        src_node_index = edges_src_index[idx];

        send_message_for_edge_iteration_cuda(node_states, src_node_index, idx, joint_probabilities, current_edge_messages);
    }
}

/**
 * Combines messages on the edge
 * @param edge_index The current edge's index
 * @param current_messages The current message to combine
 * @param dest_node_index The destination node's index in the graph
 * @param belief The current belief being buffered
 * @param num_variables The number of states within the belief
 */
__device__
void combine_loopy_edge_cuda(unsigned int edge_index, struct belief *current_messages, unsigned int dest_node_index,
                             struct belief *belief){
    unsigned int i, num_variables;
    unsigned int * address_as_uint;
    unsigned int old, assumed;
    __shared__ float current_message_value[BLOCK_SIZE], current_belief_value[BLOCK_SIZE];

    address_as_uint = (unsigned int *)current_messages;
    num_variables = current_messages[edge_index].size;

    for(i = 0; i < num_variables; ++i){
        current_message_value[threadIdx.x] = current_messages[edge_index].data[i];
        current_belief_value[threadIdx.x] = belief[dest_node_index].data[i];
        if(current_belief_value[threadIdx.x] > 0.0f){
            old = __float_as_uint(current_message_value[threadIdx.x]);
            do{
                assumed = old;
                old = atomicCAS(address_as_uint, assumed, __float_as_uint(current_belief_value[threadIdx.x] * __uint_as_float(assumed)));
            }while(assumed != old);
            belief[dest_node_index].data[i] = current_belief_value[threadIdx.x];
        }
        __syncthreads();
    }
}

/**
 * Combines incoming messages on the edge
 * @param num_edges The number of the edges in thr graph
 * @param edges_dest_index The index of the destination nodes in the graph
 * @param current_edge_messages The current edge message used for buffering
 * @param node_states The current beliefs of all nodes in the graph
 */
__global__
void combine_loopy_edge_cuda_kernel(unsigned int num_edges, unsigned int * edges_dest_index,
                                    struct belief *current_edge_messages, struct belief *node_states){
    unsigned idx, dest_node_index;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_edges; idx += blockDim.x * gridDim.x){
        dest_node_index = edges_dest_index[idx];

        combine_loopy_edge_cuda(idx, current_edge_messages, dest_node_index, node_states);
    }
}

__global__
void combine_loopy_edge_cuda_work_queue_kernel(unsigned int num_edges, unsigned int * edges_dest_index,
                                    struct belief *current_edge_messages, struct belief *node_states,
                                               unsigned int * work_queue_edges, unsigned int * num_work_items,
                                               unsigned int * work_queue_scratch, unsigned int *ctr){
    unsigned i, idx, dest_node_index;

    for(i = blockIdx.x * blockDim.x + threadIdx.x; i < *num_work_items; i += blockDim.x * gridDim.x){
        idx = work_queue_edges[i];

        dest_node_index = edges_dest_index[idx];

        combine_loopy_edge_cuda(idx, current_edge_messages, dest_node_index, node_states);
    }

    __syncthreads();
    update_work_queue_edges_cuda(work_queue_edges, num_work_items, work_queue_scratch, current_edge_messages, num_edges, PRECISION_ITERATION, ctr);
}

/**
 * Marginalizes and normalizes a belief in the graph
 * @param belief The current belief
 * @param num_vertices The number of nodes in the graph
 */
__global__
void marginalize_loop_node_edge_kernel(struct belief *belief, unsigned int num_vertices){
    unsigned int i, idx, num_variables;
    float sum;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_vertices; idx += blockDim.x * gridDim.x){
        num_variables = belief->size;
        sum = 0.0f;
        for(i = 0; i < num_variables; ++i){
            sum += belief->data[i];
        }
        if(sum >= 0.0f){
            for(i = 0; i < num_variables; ++i){
                belief->data[i] /= sum;
            }
        }
    }
}

/**
 * Calculates the delta between the current beliefs and the previous ones
 * @param i The current index of the edge
 * @param current_messages The current messages
 * @return The summed delta
 */
__device__
float calculate_local_delta(unsigned int i, struct belief * current_messages){
    float delta, diff;

    diff = current_messages[i].previous - current_messages[i].current;
    if(diff != diff){
        diff = 0.0;
    }
    delta = (float)fabs(diff);

    return delta;
}

/**
 * Calculates the delta across all messages to test for convergence via parallel reduction
 * @param current_messages The current states
 * @param delta The delta to write
 * @param delta_array Temp array to hold the partial deltas so that they can be reduced
 * @param num_edges The number of edges in the graph
 */
__global__
void calculate_delta(struct belief *current_messages,
                     float * delta, float * delta_array,
                     unsigned int num_edges){
    extern __shared__ float shared_delta[];
    unsigned int tid, idx, i, s;

    tid = threadIdx.x;
    i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_edges; idx += blockDim.x * gridDim.x){
        delta_array[idx] = calculate_local_delta(idx, current_messages);
    }
    __syncthreads();

    float my_delta = (i < num_edges) ? delta_array[i] : 0;

    if(i + BLOCK_SIZE < num_edges){
        my_delta += delta_array[i + BLOCK_SIZE];
    }

    shared_delta[tid] = my_delta;
    __syncthreads();

    // do reduction in shared memory
    for(s= blockDim.x / 2; s > 32; s>>=1){
        if(tid < s){
            shared_delta[tid] = my_delta = my_delta + shared_delta[tid + s];
        }

        __syncthreads();
    }

#if (__CUDA_ARCH__ >= 300)
    if(tid < 32){
        //fetch final intermediate sum from second warp
        if(BLOCK_SIZE >= 64){
            my_delta += shared_delta[tid + 32];
        }
        for(s = WARP_SIZE/2; s > 0; s /= 2){
            my_delta += __shfl_down(my_delta, s);
        }
    }
#else
    if((BLOCK_SIZE >= 64) && (tid < 32)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 32];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 32) && (tid < 16)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 16];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 16) && (tid < 8)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 8];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 8) && (tid < 4)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 4];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 4) && (tid < 2)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 2];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 2) && (tid < 1)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 1];
    }
    __syncthreads();
#endif
    if(tid == 0) {
        *delta = my_delta;
    }
}

/**
 * @brief Calculates the delta across all messages to test for convergence via parallel reduction
 * @details Optimized parallel reduction code borrowed from the CUDA toolkit samples
 * @param current_messages The current states
 * @param delta The delta to write
 * @param delta_array Temp array to hold partial deltas for reduction
 * @param num_edges The number of edges in the graph
 * @param n_is_pow_2 Flag to address padding for shared memory
 * @param warp_size The size of the warp of the GPU
 */
__global__
void calculate_delta_6( struct belief * current_messages,
                       float * delta, float * delta_array,
                       unsigned int num_edges, char n_is_pow_2, unsigned int warp_size) {
    extern __shared__ float shared_delta[];

    unsigned int offset;
    // perform first level of reduce
    // reading from global memory, writing to shared memory
    unsigned int idx;
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int grid_size = blockDim.x * 2 * gridDim.x;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_edges; idx += blockDim.x * gridDim.x){
        delta_array[idx] = calculate_local_delta(idx, current_messages);
    }
    __syncthreads();

    float my_delta = 0.0;

    while (i < num_edges) {
        my_delta += delta_array[i];

        // ensure we don't read out of bounds
        if (n_is_pow_2 || i + blockDim.x < num_edges) {
            my_delta += delta_array[i];
        }

        i += grid_size;
    }

    //each thread puts its local sum into shared memory
    shared_delta[tid] = my_delta;
    __syncthreads();

    // do reduction in shared mem
    if ((blockDim.x >= 512) && (tid < 256)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 256];
    }
    __syncthreads();
    if ((blockDim.x >= 256) && (tid < 128)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 128];
    }
    __syncthreads();
    if ((blockDim.x >= 128) && (tid < 64)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 64];
    }
    __syncthreads();

#if (__CUDA_ARCH__ >= 300)
    if( tid < 32){
        // fetch final intermediate sum from 2nd warp
        if(blockDim.x >= 64){
            my_delta += shared_delta[tid + 32];
        }
        for(offset = warp_size/2; offset > 0; offset /= 2 ){
            my_delta += __shfl_down(my_delta, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockDim.x >= 64) && (tid < 32)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 32];
    }
    __syncthreads();

    if ((blockDim.x >= 32) && (tid < 16)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 16];
    }
    __syncthreads();

    if((blockDim.x >= 16) && (tid < 8)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 8];
    }
    __syncthreads();

    if((blockDim.x >= 8) && (tid < 4)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 4];
    }
    __syncthreads();

    if((blockDim.x >= 4) && (tid < 2)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 2];
    }
    __syncthreads();

    if((blockDim.x >= 2) && (tid < 1)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 1];
    }
    __syncthreads();

#endif
    //write result for this block to global mem
    if(tid == 0){
        *delta = my_delta;
    }
}

/**
 * Calculates the delta across all messages to test for convergence via parallel reduction
 * @details Simple implementation used for comparison against reduction code
 * @param current_messages The current messages
 * @param delta The delta to write
 * @param delta_array Temp array to hold partial deltas to be used for reduction
 * @param num_edges The number of the edges in the graph
 */
__global__
void calculate_delta_simple(struct belief * current_messages,
                            float * delta, float * delta_array,
                            unsigned int num_edges) {
    extern __shared__ float shared_delta[];
    unsigned int tid, idx, i, s;

    tid = threadIdx.x;
    idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_edges; idx += blockDim.x * gridDim.x){
        delta_array[idx] = calculate_local_delta(idx, current_messages);
    }
    __syncthreads();

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    shared_delta[tid] = (idx < num_edges) ? delta_array[idx] : 0;

    __syncthreads();

    // do reduction in shared mem
    for(s = 1; s < blockDim.x; s *= 2){
        i = 2 * s * tid;
        if( i < blockDim.x ) {
            shared_delta[i] += shared_delta[i + s];
        }

        __syncthreads();
    }

    //write result for this block to global mem
    if(tid == 0){
        *delta = shared_delta[0];
    }
}

__global__
void marginalize_viterbi_beliefs(struct belief * nodes, unsigned int num_nodes){
    unsigned int idx, i, num_variables;
    float sum;

    for(idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_nodes; idx += blockDim.x * gridDim.x){
        sum = 0.0;
        for(i = 0; i < nodes[idx].size; ++i){
            sum += nodes[idx].data[i];
        }
        for(i = 0; i < nodes[idx].size; ++i){
            nodes[idx].data[i] = nodes[idx].data[i] / sum;
        }
    }
}

/**
 * Helper function to test for error with CUDA kernel execution
 */
void test_error(){
    hipError_t err;

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

/**
 * Runs loopy BP on the GPU
 * @param graph The graph to run
 * @param convergence The convergence threshold; when the delta falls below this threshold, execution will halt
 * @param max_iterations The number of executions to stop at
 * @return The actual number of iterations ran
 */
unsigned int loopy_propagate_until_cuda(Graph_t graph, float convergence, unsigned int max_iterations){
    unsigned int i, j, num_iter, num_vertices, num_edges;
    float * delta;
    float * delta_array;
    float previous_delta, host_delta;
    char is_pow_2;
    unsigned int *ctr;

    struct joint_probability * edges_joint_probabilities;

    struct belief * current_messages;

    struct belief * node_states;

    host_delta = 0.0;

    init_work_queue_nodes(graph);

    struct hipChannelFormatDesc channel_desc_unsigned_int = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    unsigned int * dest_node_to_edges_nodes;
    unsigned int * dest_node_to_edges_edges;
    unsigned int * src_node_to_edges_nodes;
    unsigned int * src_node_to_edges_edges;
    unsigned int * work_queue_nodes;
    unsigned int * work_queue_scratch;
    unsigned int * num_work_items;

    num_vertices = graph->current_num_vertices;
    num_edges = graph->current_num_edges;

    /*printf("Before=====");
    print_edges(graph);
    print_nodes(graph);*/


    is_pow_2 = num_vertices % 2 == 0;

    // allocate data
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&current_messages, sizeof(struct belief) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_states, sizeof(struct belief) * graph->current_num_vertices));

    CUDA_CHECK_RETURN(hipMalloc((void **)&work_queue_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&work_queue_scratch, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&num_work_items, sizeof(unsigned int)));

    CUDA_CHECK_RETURN(hipMalloc((void **)&delta, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&delta_array, sizeof(float) * num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&ctr, sizeof(unsigned int)));


    // copy data
    CUDA_CHECK_RETURN(hipMemcpy(edges_joint_probabilities, graph->edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges, hipMemcpyHostToDevice ));

    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->edges_messages, sizeof(struct belief) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(node_states, graph->node_states, sizeof(struct belief) *  graph->current_num_vertices, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges_nodes, graph->dest_nodes_to_edges_node_list, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges_edges, graph->dest_nodes_to_edges_edge_list, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges_nodes, graph->src_nodes_to_edges_node_list, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges_edges, graph->src_nodes_to_edges_edge_list, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(work_queue_nodes, graph->work_queue_nodes, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(num_work_items, &graph->num_work_items_nodes, sizeof(unsigned int), hipMemcpyHostToDevice));

    const int edgeCount = (num_edges + BLOCK_SIZE - 1)/ BLOCK_SIZE;
    const int nodeCount = (num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;
    num_iter = 0;

    dim3 dimReduceBlock(BLOCK_SIZE, 1, 1);
    dim3 dimReduceGrid(edgeCount, 1, 1);
    int reduceSmemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

    for(i = 0; i < max_iterations; i+= BATCH_SIZE){
        for(j = 0; j < BATCH_SIZE; ++j) {
            CUDA_CHECK_RETURN(hipMemset(ctr, 0, sizeof(unsigned int)));

            loopy_propagate_main_loop<<<nodeCount, BLOCK_SIZE >>>(num_vertices, num_edges,
            node_states,
            edges_joint_probabilities,
            current_messages,
            work_queue_nodes, num_work_items,
            work_queue_scratch,
            src_node_to_edges_nodes, src_node_to_edges_edges,
            dest_node_to_edges_nodes, dest_node_to_edges_edges, ctr);
            test_error();
            num_iter++;
        }
        calculate_delta_6<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges, is_pow_2, WARP_SIZE);
        //calculate_delta<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        //calculate_delta_simple<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        test_error();
        CUDA_CHECK_RETURN(hipMemcpy(&host_delta, delta, sizeof(float), hipMemcpyDeviceToHost));
        //printf("Current delta: %f\n", host_delta);

        if(host_delta < convergence || fabs(host_delta - previous_delta) < convergence){
            break;
        }
        previous_delta = host_delta;
    }

    // copy data back
    CUDA_CHECK_RETURN(hipMemcpy(graph->node_states, node_states, sizeof(struct belief) * num_vertices, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->edges_messages, current_messages, sizeof(struct belief) * num_edges, hipMemcpyDeviceToHost));

    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges_edges));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges_edges));

    CUDA_CHECK_RETURN(hipFree(edges_joint_probabilities));

    CUDA_CHECK_RETURN(hipFree(current_messages));

    CUDA_CHECK_RETURN(hipFree(node_states));

    CUDA_CHECK_RETURN(hipFree(delta));
    CUDA_CHECK_RETURN(hipFree(delta_array));

    CUDA_CHECK_RETURN(hipFree(ctr));

    CUDA_CHECK_RETURN(hipFree(work_queue_nodes));
    CUDA_CHECK_RETURN(hipFree(work_queue_scratch));
    CUDA_CHECK_RETURN(hipFree(num_work_items));

    /*printf("After=====");
    print_nodes(graph);
    print_edges(graph);*/

    return num_iter;
}

static void *launch_read_kernel(void *data) {
    struct stream_data *stream_data;

    stream_data = (struct stream_data *)data;
    const int nodeCount = (stream_data->num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;

    loopy_propagate_partitioned_read_main_loop<<<nodeCount, BLOCK_SIZE >>>(stream_data->num_vertices,
    stream_data->num_edges,
    stream_data->node_messages,
    stream_data->current_edge_messages,
    stream_data->work_queue_nodes, stream_data->num_work_items,
    stream_data->dest_nodes_to_edges_nodes, stream_data->dest_nodes_to_edges_edges,
    stream_data->node_buffer,
    stream_data->nodes_to_partitions, stream_data->current_partition);

    hipStreamSynchronize(0);

    return NULL;
}

static void *launch_send_kernel(void *data) {
    struct stream_data *stream_data;

    stream_data = (struct stream_data *)data;
    const int nodeCount = (stream_data->num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;

    loopy_propagate_partitioned_send_main_loop<<<nodeCount, BLOCK_SIZE >>>(stream_data->num_vertices,
    stream_data->num_edges,
    stream_data->node_messages,
    stream_data->joint_probabilities,
    stream_data->current_edge_messages,
    stream_data->work_queue_nodes, stream_data->num_work_items,
    stream_data->work_queue_scratch,
    stream_data->src_nodes_to_edges_nodes, stream_data->src_nodes_to_edges_edges,
    stream_data->dest_nodes_to_edges_nodes, stream_data->dest_nodes_to_edges_edges,
    stream_data->node_buffer,
    stream_data->nodes_to_partitions, stream_data->current_partition);

    hipStreamSynchronize(0);

    return NULL;
}

unsigned int loopy_propagate_until_cuda_partitioned(Graph_t graph, float convergence, unsigned int max_iterations,
                                                    unsigned int num_partitions){
    unsigned int i, j, k, num_iter, num_vertices, num_edges;
    float * delta;
    float * delta_array;
    float previous_delta, host_delta;
    char is_pow_2;

    struct joint_probability * edges_joint_probabilities;

    struct belief * current_messages;

    struct belief * node_states;
    struct belief * node_buffer;

    unsigned int *ctr;

    host_delta = 0.0;

    init_work_queue_nodes(graph);

    struct hipChannelFormatDesc channel_desc_unsigned_int = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    unsigned int * dest_node_to_edges_nodes;
    unsigned int * dest_node_to_edges_edges;
    unsigned int * src_node_to_edges_nodes;
    unsigned int * src_node_to_edges_edges;
    unsigned int * work_queue_nodes;
    unsigned int * work_queue_scratch;
    unsigned int * num_work_items;
    idx_t * nodes_to_partitions;

    pthread_t threads[num_partitions];
    hipStream_t streams[num_partitions];
    struct stream_data thread_data[num_partitions];




    num_vertices = graph->current_num_vertices;
    num_edges = graph->current_num_edges;

    /*printf("Before=====");
    print_edges(graph);
    print_nodes(graph);*/


    is_pow_2 = num_vertices % 2 == 0;

    // allocate data
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&current_messages, sizeof(struct belief) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_states, sizeof(struct belief) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_buffer, sizeof(struct belief) * graph->current_num_vertices));

    CUDA_CHECK_RETURN(hipMalloc((void **)&nodes_to_partitions, sizeof(idx_t) * graph->current_num_vertices));

    CUDA_CHECK_RETURN(hipMalloc((void **)&work_queue_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&work_queue_scratch, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&num_work_items, sizeof(unsigned int)));

    CUDA_CHECK_RETURN(hipMalloc((void **)&delta, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&delta_array, sizeof(float) * num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&ctr, sizeof(unsigned int)));


    // copy data
    CUDA_CHECK_RETURN(hipMemcpy(edges_joint_probabilities, graph->edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges, hipMemcpyHostToDevice ));

    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->edges_messages, sizeof(struct belief) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(node_states, graph->node_states, sizeof(struct belief) *  graph->current_num_vertices, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(nodes_to_partitions, graph->partitioned_nodes, sizeof(idx_t) *  graph->current_num_vertices, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges_nodes, graph->dest_nodes_to_edges_node_list, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges_edges, graph->dest_nodes_to_edges_edge_list, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges_nodes, graph->src_nodes_to_edges_node_list, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges_edges, graph->src_nodes_to_edges_edge_list, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(work_queue_nodes, graph->work_queue_nodes, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(num_work_items, &graph->num_work_items_nodes, sizeof(unsigned int), hipMemcpyHostToDevice));

    const int edgeCount = (num_edges + BLOCK_SIZE - 1)/ BLOCK_SIZE;
    const int nodeCount = (num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;
    num_iter = 0;

    dim3 dimReduceBlock(BLOCK_SIZE, 1, 1);
    dim3 dimReduceGrid(edgeCount, 1, 1);
    int reduceSmemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

    // prepare streams and data
    for(i = 0; i < num_partitions; ++i) {
        hipStreamCreate(&streams[i]);

        thread_data[i].num_vertices = num_vertices;
        thread_data[i].num_edges = num_edges;
        thread_data[i].node_messages = node_states;
        thread_data[i].joint_probabilities = edges_joint_probabilities;
        thread_data[i].current_edge_messages = current_messages;
        thread_data[i].work_queue_nodes = work_queue_nodes;
        thread_data[i].num_work_items = num_work_items;
        thread_data[i].work_queue_scratch = work_queue_scratch;
        thread_data[i].src_nodes_to_edges_nodes = src_node_to_edges_nodes;
        thread_data[i].src_nodes_to_edges_edges = src_node_to_edges_edges;
        thread_data[i].dest_nodes_to_edges_nodes = dest_node_to_edges_nodes;
        thread_data[i].dest_nodes_to_edges_edges = dest_node_to_edges_edges;
        thread_data[i].node_buffer = node_buffer;
        thread_data[i].nodes_to_partitions = nodes_to_partitions;
        thread_data[i].current_partition = (idx_t)i;
    }

    for(i = 0; i < max_iterations; i+= BATCH_SIZE){
        for(j = 0; j < BATCH_SIZE; ++j) {
            hipMemset(ctr, 0, sizeof(unsigned int));

            // read data
            for(k = 0; k < num_partitions; ++k) {
                if (pthread_create(&threads[i], NULL, launch_read_kernel, &thread_data[i])) {
                    fprintf(stderr, "Error creating read thread %d\n", k);
                    return 1;
                }
            }
            for(k = 0; k < num_partitions; ++k) {
                if (pthread_join(threads[i], NULL)) {
                    fprintf(stderr, "Error joining read thread %d\n", k);
                }
            }

            // send data
            for(k = 0; k < num_partitions; ++k) {
                if (pthread_create(&threads[i], NULL, launch_send_kernel, &thread_data[i])) {
                    fprintf(stderr, "Error creating send thread %d\n", k);
                }
            }
            for(k = 0; k < num_partitions; ++k) {
                if (pthread_join(threads[i], NULL)) {
                    fprintf(stderr, "Error joining send thread %d\n", k);
                }
            }

            // update
            loopy_propagate_partitioned_update<<<nodeCount, BLOCK_SIZE>>>(num_vertices, node_states, work_queue_nodes,
                    num_work_items, work_queue_scratch, ctr);

            test_error();
            num_iter++;
        }
        calculate_delta_6<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges, is_pow_2, WARP_SIZE);
        //calculate_delta<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        //calculate_delta_simple<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        test_error();
        CUDA_CHECK_RETURN(hipMemcpy(&host_delta, delta, sizeof(float), hipMemcpyDeviceToHost));
        //printf("Current delta: %f\n", host_delta);

        if(host_delta < convergence || fabs(host_delta - previous_delta) < convergence){
            break;
        }
        previous_delta = host_delta;
    }

    // copy data back
    CUDA_CHECK_RETURN(hipMemcpy(graph->node_states, node_states, sizeof(struct belief) * num_vertices, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->edges_messages, current_messages, sizeof(struct belief) * num_edges, hipMemcpyDeviceToHost));

    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges_edges));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges_edges));

    CUDA_CHECK_RETURN(hipFree(edges_joint_probabilities));

    CUDA_CHECK_RETURN(hipFree(current_messages));

    CUDA_CHECK_RETURN(hipFree(node_states));
    CUDA_CHECK_RETURN(hipFree(node_buffer));

    CUDA_CHECK_RETURN(hipFree(nodes_to_partitions));

    CUDA_CHECK_RETURN(hipFree(delta));
    CUDA_CHECK_RETURN(hipFree(delta_array));
    CUDA_CHECK_RETURN(hipFree(ctr));

    CUDA_CHECK_RETURN(hipFree(work_queue_nodes));
    CUDA_CHECK_RETURN(hipFree(work_queue_scratch));
    CUDA_CHECK_RETURN(hipFree(num_work_items));

    /*printf("After=====");
    print_nodes(graph);
    print_edges(graph);*/

    return num_iter;
}


/**
 * Runs PageRank on the GPU
 * @param graph The graph to run
 * @param convergence The convergence threshold; when the delta falls below this threshold, execution will halt
 * @param max_iterations The number of executions to stop at
 * @return The actual number of iterations ran
 */
unsigned int page_rank_until_cuda(Graph_t graph, float convergence, unsigned int max_iterations){
    unsigned int i, j, num_iter, num_vertices, num_edges;
    float * delta;
    float * delta_array;
    float previous_delta, host_delta;
    char is_pow_2;

    struct joint_probability * edges_joint_probabilities;

    struct belief * current_messages;

    struct belief * node_states;

    host_delta = 0.0;

    struct hipChannelFormatDesc channel_desc_unsigned_int = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    unsigned int * dest_node_to_edges_nodes;
    unsigned int * dest_node_to_edges_edges;
    unsigned int * src_node_to_edges_nodes;
    unsigned int * src_node_to_edges_edges;

    num_vertices = graph->current_num_vertices;
    num_edges = graph->current_num_edges;

    /*printf("Before=====");
    print_edges(graph);
    print_nodes(graph);*/


    is_pow_2 = num_vertices % 2 == 0;

    // allocate data
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&current_messages, sizeof(struct belief) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_states, sizeof(struct belief) * graph->current_num_vertices));

    CUDA_CHECK_RETURN(hipMalloc((void **)&delta, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&delta_array, sizeof(float) * num_edges));


    // copy data
    CUDA_CHECK_RETURN(hipMemcpy(edges_joint_probabilities, graph->edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges, hipMemcpyHostToDevice ));

    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->edges_messages, sizeof(struct belief) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(node_states, graph->node_states, sizeof(struct belief) *  graph->current_num_vertices, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges_nodes, graph->dest_nodes_to_edges_node_list, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges_edges, graph->dest_nodes_to_edges_edge_list, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges_nodes, graph->src_nodes_to_edges_node_list, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges_edges, graph->src_nodes_to_edges_edge_list, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));

    const int edgeCount = (num_edges + BLOCK_SIZE - 1)/ BLOCK_SIZE;
    const int nodeCount = (num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;
    num_iter = 0;

    dim3 dimReduceBlock(BLOCK_SIZE, 1, 1);
    dim3 dimReduceGrid(edgeCount, 1, 1);
    int reduceSmemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

    for(i = 0; i < max_iterations; i+= BATCH_SIZE){
        for(j = 0; j < BATCH_SIZE; ++j) {
            page_rank_main_loop<<<nodeCount, BLOCK_SIZE >>>(num_vertices, num_edges, node_states, edges_joint_probabilities, current_messages, src_node_to_edges_nodes, src_node_to_edges_edges, src_node_to_edges_nodes, src_node_to_edges_edges);
            test_error();
            num_iter++;
        }
        calculate_delta_6<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges, is_pow_2, WARP_SIZE);
        //calculate_delta<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        //calculate_delta_simple<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        test_error();
        CUDA_CHECK_RETURN(hipMemcpy(&host_delta, delta, sizeof(float), hipMemcpyDeviceToHost));
        //printf("Current delta: %f\n", host_delta);

        if(host_delta < convergence || fabs(host_delta - previous_delta) < convergence){
            break;
        }
        previous_delta = host_delta;
    }

    // copy data back
    CUDA_CHECK_RETURN(hipMemcpy(graph->node_states, node_states, sizeof(struct belief) * num_vertices, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->edges_messages, current_messages, sizeof(struct belief) * num_edges, hipMemcpyDeviceToHost));

    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges_edges));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges_edges));

    CUDA_CHECK_RETURN(hipFree(edges_joint_probabilities));

    CUDA_CHECK_RETURN(hipFree(current_messages));

    CUDA_CHECK_RETURN(hipFree(node_states));

    CUDA_CHECK_RETURN(hipFree(delta));
    CUDA_CHECK_RETURN(hipFree(delta_array));

    /*printf("After=====");
    print_nodes(graph);
    print_edges(graph);*/

    return num_iter;
}


/**
 * Runs Viterbi on the GPU
 * @param graph The graph to run
 * @param convergence The convergence threshold; when the delta falls below this threshold, execution will halt
 * @param max_iterations The number of executions to stop at
 * @return The actual number of iterations ran
 */
unsigned int viterbi_until_cuda(Graph_t graph, float convergence, unsigned int max_iterations){
    unsigned int i, j, num_iter, num_vertices, num_edges;
    float * delta;
    float * delta_array;
    float previous_delta, host_delta;
    char is_pow_2;

    struct joint_probability * edges_joint_probabilities;

    struct belief * current_messages;

    struct belief * node_states;

    host_delta = 0.0;

    struct hipChannelFormatDesc channel_desc_unsigned_int = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    unsigned int * dest_node_to_edges_nodes;
    unsigned int * dest_node_to_edges_edges;
    unsigned int * src_node_to_edges_nodes;
    unsigned int * src_node_to_edges_edges;

    num_vertices = graph->current_num_vertices;
    num_edges = graph->current_num_edges;

    /*printf("Before=====");
    print_edges(graph);
    print_nodes(graph);*/


    is_pow_2 = num_vertices % 2 == 0;

    // allocate data
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&current_messages, sizeof(struct belief) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_states, sizeof(struct belief) * graph->current_num_vertices));

    CUDA_CHECK_RETURN(hipMalloc((void **)&delta, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&delta_array, sizeof(float) * num_edges));


    // copy data
    CUDA_CHECK_RETURN(hipMemcpy(edges_joint_probabilities, graph->edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges, hipMemcpyHostToDevice ));

    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->edges_messages, sizeof(struct belief) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(node_states, graph->node_states, sizeof(struct belief) *  graph->current_num_vertices, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges_nodes, graph->dest_nodes_to_edges_node_list, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges_edges, graph->dest_nodes_to_edges_edge_list, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges_nodes, graph->src_nodes_to_edges_node_list, sizeof(unsigned int) * num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges_edges, graph->src_nodes_to_edges_edge_list, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));

    const int edgeCount = (num_edges + BLOCK_SIZE - 1)/ BLOCK_SIZE;
    const int nodeCount = (num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;
    num_iter = 0;

    dim3 dimReduceBlock(BLOCK_SIZE, 1, 1);
    dim3 dimReduceGrid(edgeCount, 1, 1);
    int reduceSmemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

    for(i = 0; i < max_iterations; i+= BATCH_SIZE){
        for(j = 0; j < BATCH_SIZE; ++j) {
            viterbi_main_loop<<<nodeCount, BLOCK_SIZE >>>(num_vertices, num_edges, node_states, edges_joint_probabilities, current_messages, src_node_to_edges_nodes, src_node_to_edges_edges, src_node_to_edges_nodes, src_node_to_edges_edges);
            test_error();
            num_iter++;
        }
        calculate_delta_6<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges, is_pow_2, WARP_SIZE);
        //calculate_delta<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        //calculate_delta_simple<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        test_error();
        CUDA_CHECK_RETURN(hipMemcpy(&host_delta, delta, sizeof(float), hipMemcpyDeviceToHost));
        //printf("Current delta: %f\n", host_delta);

        if(host_delta < convergence || fabs(host_delta - previous_delta) < convergence){
            marginalize_viterbi_beliefs<<<nodeCount, BLOCK_SIZE >>>(node_states, num_vertices);
            break;
        }
        previous_delta = host_delta;
    }

    // copy data back
    CUDA_CHECK_RETURN(hipMemcpy(graph->node_states, node_states, sizeof(struct belief) * num_vertices, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->edges_messages, current_messages, sizeof(struct belief) * num_edges, hipMemcpyDeviceToHost));

    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges_edges));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges_edges));

    CUDA_CHECK_RETURN(hipFree(edges_joint_probabilities));

    CUDA_CHECK_RETURN(hipFree(current_messages));

    CUDA_CHECK_RETURN(hipFree(node_states));

    CUDA_CHECK_RETURN(hipFree(delta));
    CUDA_CHECK_RETURN(hipFree(delta_array));

    /*printf("After=====");
    print_nodes(graph);
    print_edges(graph);*/

    return num_iter;
}

/**
 * Runs the edge-optimized loopy BP code
 * @param graph The graph to use
 * @param convergence The convergence threshold; when the delta falls below this threshold, execution will stop
 * @param max_iterations The maximum number of iterations to run for
 * @return The actual number of iterations ran
 */
unsigned int loopy_propagate_until_cuda_edge(Graph_t graph, float convergence, unsigned int max_iterations){
    unsigned int i, j, num_iter, num_vertices, num_edges;
    float * delta;
    float * delta_array;
    float previous_delta, host_delta;
    char is_pow_2;

    unsigned int *ctr;

    struct joint_probability * edges_joint_probabilities;
    struct belief * current_messages;
    struct belief * node_states;

    unsigned int * edges_src_index;
    unsigned int * edges_dest_index;
    unsigned int * dest_nodes_to_edges_nodes;
    unsigned int * dest_nodes_to_edges_edges;

    unsigned int * work_queue_edges;
    unsigned int * work_queue_scratch;
    unsigned int * num_work_items;

    init_work_queue_edges(graph);

    host_delta = 0.0;

    struct hipChannelFormatDesc channel_desc_unsigned_int = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    num_vertices = graph->current_num_vertices;
    num_edges = graph->current_num_edges;

    /*printf("Before=====");
    print_edges(graph);
    print_nodes(graph);*/


    is_pow_2 = num_vertices % 2 == 0;

    // allocate data
    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_src_index, sizeof(unsigned int) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_dest_index, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_states, sizeof(struct belief) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&current_messages, sizeof(struct belief) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_nodes_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_nodes_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&delta, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&delta_array, sizeof(float) * num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&ctr, sizeof(unsigned int)));

    CUDA_CHECK_RETURN(hipMalloc((void **)&work_queue_edges, sizeof(unsigned int) * num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&work_queue_scratch, sizeof(unsigned int) * num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&num_work_items, sizeof(unsigned int)));


    // copy data
    CUDA_CHECK_RETURN(hipMemcpy(edges_joint_probabilities, graph->edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges, hipMemcpyHostToDevice ));
    CUDA_CHECK_RETURN(hipMemcpy(node_states, graph->node_states, sizeof(struct belief) * graph->current_num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->edges_messages, sizeof(struct belief) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(edges_src_index, graph->edges_src_index, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(edges_dest_index, graph->edges_dest_index, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(dest_nodes_to_edges_nodes, graph->dest_nodes_to_edges_node_list, sizeof(unsigned int) * graph->current_num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dest_nodes_to_edges_edges, graph->dest_nodes_to_edges_edge_list, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(work_queue_edges, graph->work_queue_edges, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(num_work_items, &num_edges, sizeof(unsigned int), hipMemcpyHostToDevice));

    const int edgeCount = (num_edges + BLOCK_SIZE - 1)/ BLOCK_SIZE;
    const int nodeCount = (num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;
    num_iter = 0;

    dim3 dimReduceBlock(BLOCK_SIZE, 1, 1);
    dim3 dimReduceGrid(edgeCount, 1, 1);
    int reduceSmemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

    for(i = 0; i < max_iterations; i+= BATCH_SIZE){
        for(j = 0; j < BATCH_SIZE; ++j) {
            hipMemset(ctr, 0, sizeof(unsigned int));

            send_message_for_edge_iteration_cuda_work_queue_kernel<<<edgeCount, BLOCK_SIZE >>>(num_edges, edges_src_index, node_states, edges_joint_probabilities, current_messages, work_queue_edges, num_work_items);
            test_error();
            combine_loopy_edge_cuda_work_queue_kernel<<<edgeCount, BLOCK_SIZE>>>(num_edges, edges_dest_index, current_messages, node_states, work_queue_edges, num_work_items, work_queue_scratch, ctr);
            test_error();
            //marginalize_loop_node_edge_kernel<<<nodeCount, BLOCK_SIZE>>>(node_states, num_vars, num_vertices);
            marginalize_nodes<<<nodeCount, BLOCK_SIZE>>>(node_states, current_messages,
            dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
            test_error();

            num_iter++;
        }
        calculate_delta_6<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges, is_pow_2, WARP_SIZE);
        //calculate_delta<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        //calculate_delta_simple<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        test_error();
        CUDA_CHECK_RETURN(hipMemcpy(&host_delta, delta, sizeof(float), hipMemcpyDeviceToHost));
        //   printf("Current delta: %f\n", host_delta);

        if(host_delta < convergence || fabs(host_delta - previous_delta) < convergence){
            break;
        }
        previous_delta = host_delta;
    }

    // copy data back
    CUDA_CHECK_RETURN(hipMemcpy(graph->node_states, node_states, sizeof(struct belief) * num_vertices, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->edges_messages, current_messages, sizeof(struct belief) * num_edges, hipMemcpyDeviceToHost));

    CUDA_CHECK_RETURN(hipFree(edges_joint_probabilities));
    CUDA_CHECK_RETURN(hipFree(current_messages));
    CUDA_CHECK_RETURN(hipFree(node_states));

    CUDA_CHECK_RETURN(hipFree(edges_src_index));
    CUDA_CHECK_RETURN(hipFree(edges_dest_index));

    CUDA_CHECK_RETURN(hipFree(dest_nodes_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(dest_nodes_to_edges_edges));

    CUDA_CHECK_RETURN(hipFree(delta));
    CUDA_CHECK_RETURN(hipFree(delta_array));
    CUDA_CHECK_RETURN(hipFree(ctr));

    CUDA_CHECK_RETURN(hipFree(work_queue_edges));
    CUDA_CHECK_RETURN(hipFree(work_queue_scratch));
    CUDA_CHECK_RETURN(hipFree(num_work_items));

    /*printf("After=====");
    print_nodes(graph);
    print_edges(graph);*/

    return num_iter;
}


/**
 * Runs the edge-optimized PageRank code
 * @param graph The graph to use
 * @param convergence The convergence threshold; when the delta falls below this threshold, execution will stop
 * @param max_iterations The maximum number of iterations to run for
 * @return The actual number of iterations ran
 */
unsigned int page_rank_until_cuda_edge(Graph_t graph, float convergence, unsigned int max_iterations){
    unsigned int i, j, num_iter, num_vertices, num_edges;
    float * delta;
    float * delta_array;
    float previous_delta, host_delta;
    char is_pow_2;

    struct joint_probability * edges_joint_probabilities;
    struct belief * current_messages;
    struct belief * node_states;

    unsigned int * edges_src_index;
    unsigned int * edges_dest_index;
    unsigned int * dest_nodes_to_edges_nodes;
    unsigned int * dest_nodes_to_edges_edges;

    host_delta = 0.0;

    struct hipChannelFormatDesc channel_desc_unsigned_int = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    num_vertices = graph->current_num_vertices;
    num_edges = graph->current_num_edges;

    /*printf("Before=====");
    print_edges(graph);
    print_nodes(graph);*/


    is_pow_2 = num_vertices % 2 == 0;

    // allocate data
    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_src_index, sizeof(unsigned int) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_dest_index, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_states, sizeof(struct belief) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&current_messages, sizeof(struct belief) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_nodes_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_nodes_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&delta, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&delta_array, sizeof(float) * num_edges));


    // copy data
    CUDA_CHECK_RETURN(hipMemcpy(edges_joint_probabilities, graph->edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges, hipMemcpyHostToDevice ));
    CUDA_CHECK_RETURN(hipMemcpy(node_states, graph->node_states, sizeof(struct belief) * graph->current_num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->edges_messages, sizeof(struct belief) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(edges_src_index, graph->edges_src_index, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(edges_dest_index, graph->edges_dest_index, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(dest_nodes_to_edges_nodes, graph->dest_nodes_to_edges_node_list, sizeof(unsigned int) * graph->current_num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dest_nodes_to_edges_edges, graph->dest_nodes_to_edges_edge_list, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));

    const int edgeCount = (num_edges + BLOCK_SIZE - 1)/ BLOCK_SIZE;
    const int nodeCount = (num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;
    num_iter = 0;

    dim3 dimReduceBlock(BLOCK_SIZE, 1, 1);
    dim3 dimReduceGrid(edgeCount, 1, 1);
    int reduceSmemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

    for(i = 0; i < max_iterations; i+= BATCH_SIZE){
        for(j = 0; j < BATCH_SIZE; ++j) {
            send_message_for_edge_iteration_cuda_kernel<<<edgeCount, BLOCK_SIZE >>>(num_edges, edges_src_index, node_states, edges_joint_probabilities, current_messages);
            test_error();
            combine_loopy_edge_cuda_kernel<<<edgeCount, BLOCK_SIZE>>>(num_edges, edges_dest_index, current_messages, node_states);
            test_error();
            //marginalize_loop_node_edge_kernel<<<nodeCount, BLOCK_SIZE>>>(node_states, num_vars, num_vertices);
            marginalize_page_rank_nodes<<<nodeCount, BLOCK_SIZE>>>(node_states, current_messages,
                    dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
            test_error();

            num_iter++;
        }
        calculate_delta_6<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges, is_pow_2, WARP_SIZE);
        //calculate_delta<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        //calculate_delta_simple<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        test_error();
        CUDA_CHECK_RETURN(hipMemcpy(&host_delta, delta, sizeof(float), hipMemcpyDeviceToHost));
        //   printf("Current delta: %f\n", host_delta);

        if(host_delta < convergence || fabs(host_delta - previous_delta) < convergence){
            break;
        }
        previous_delta = host_delta;
    }

    // copy data back
    CUDA_CHECK_RETURN(hipMemcpy(graph->node_states, node_states, sizeof(struct belief) * num_vertices, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->edges_messages, current_messages, sizeof(struct belief) * num_edges, hipMemcpyDeviceToHost));

    CUDA_CHECK_RETURN(hipFree(edges_joint_probabilities));
    CUDA_CHECK_RETURN(hipFree(current_messages));
    CUDA_CHECK_RETURN(hipFree(node_states));

    CUDA_CHECK_RETURN(hipFree(edges_src_index));
    CUDA_CHECK_RETURN(hipFree(edges_dest_index));

    CUDA_CHECK_RETURN(hipFree(dest_nodes_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(dest_nodes_to_edges_edges));

    CUDA_CHECK_RETURN(hipFree(delta));
    CUDA_CHECK_RETURN(hipFree(delta_array));

    /*printf("After=====");
    print_nodes(graph);
    print_edges(graph);*/

    return num_iter;
}


/**
 * Runs the edge-optimized Viterbi code
 * @param graph The graph to use
 * @param convergence The convergence threshold; when the delta falls below this threshold, execution will stop
 * @param max_iterations The maximum number of iterations to run for
 * @return The actual number of iterations ran
 */
unsigned int viterbi_until_cuda_edge(Graph_t graph, float convergence, unsigned int max_iterations){
    unsigned int i, j, num_iter, num_vertices, num_edges;
    float * delta;
    float * delta_array;
    float previous_delta, host_delta;
    char is_pow_2;

    struct joint_probability * edges_joint_probabilities;
    struct belief * current_messages;
    struct belief * node_states;

    unsigned int * edges_src_index;
    unsigned int * edges_dest_index;
    unsigned int * dest_nodes_to_edges_nodes;
    unsigned int * dest_nodes_to_edges_edges;

    host_delta = 0.0;

    struct hipChannelFormatDesc channel_desc_unsigned_int = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    num_vertices = graph->current_num_vertices;
    num_edges = graph->current_num_edges;

    /*printf("Before=====");
    print_edges(graph);
    print_nodes(graph);*/


    is_pow_2 = num_vertices % 2 == 0;

    // allocate data
    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_src_index, sizeof(unsigned int) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_dest_index, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_states, sizeof(struct belief) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&current_messages, sizeof(struct belief) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_nodes_to_edges_nodes, sizeof(unsigned int) * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_nodes_to_edges_edges, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&delta, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&delta_array, sizeof(float) * num_edges));


    // copy data
    CUDA_CHECK_RETURN(hipMemcpy(edges_joint_probabilities, graph->edges_joint_probabilities, sizeof(struct joint_probability) * graph->current_num_edges, hipMemcpyHostToDevice ));
    CUDA_CHECK_RETURN(hipMemcpy(node_states, graph->node_states, sizeof(struct belief) * graph->current_num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->edges_messages, sizeof(struct belief) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(edges_src_index, graph->edges_src_index, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(edges_dest_index, graph->edges_dest_index, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(dest_nodes_to_edges_nodes, graph->dest_nodes_to_edges_node_list, sizeof(unsigned int) * graph->current_num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(dest_nodes_to_edges_edges, graph->dest_nodes_to_edges_edge_list, sizeof(unsigned int) * graph->current_num_edges, hipMemcpyHostToDevice));

    const int edgeCount = (num_edges + BLOCK_SIZE - 1)/ BLOCK_SIZE;
    const int nodeCount = (num_vertices + BLOCK_SIZE - 1) / BLOCK_SIZE;
    num_iter = 0;

    dim3 dimReduceBlock(BLOCK_SIZE, 1, 1);
    dim3 dimReduceGrid(edgeCount, 1, 1);
    int reduceSmemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

    for(i = 0; i < max_iterations; i+= BATCH_SIZE){
        for(j = 0; j < BATCH_SIZE; ++j) {
            send_message_for_edge_iteration_cuda_kernel<<<edgeCount, BLOCK_SIZE >>>(num_edges, edges_src_index, node_states, edges_joint_probabilities, current_messages);
            test_error();
            combine_loopy_edge_cuda_kernel<<<edgeCount, BLOCK_SIZE>>>(num_edges, edges_dest_index, current_messages, node_states);
            test_error();
            //marginalize_loop_node_edge_kernel<<<nodeCount, BLOCK_SIZE>>>(node_states, num_vars, num_vertices);
            argmax_nodes<<<nodeCount, BLOCK_SIZE>>>(node_states, current_messages,
                    dest_nodes_to_edges_nodes, dest_nodes_to_edges_edges, num_vertices, num_edges);
            test_error();

            num_iter++;
        }
        calculate_delta_6<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges, is_pow_2, WARP_SIZE);
        //calculate_delta<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        //calculate_delta_simple<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(current_messages, delta, delta_array, num_edges);
        test_error();
        CUDA_CHECK_RETURN(hipMemcpy(&host_delta, delta, sizeof(float), hipMemcpyDeviceToHost));
        //   printf("Current delta: %f\n", host_delta);

        if(host_delta < convergence || fabs(host_delta - previous_delta) < convergence){
            marginalize_viterbi_beliefs<<<nodeCount, BLOCK_SIZE >>>(node_states, num_vertices);
            break;
        }
        previous_delta = host_delta;
    }

    // copy data back
    CUDA_CHECK_RETURN(hipMemcpy(graph->node_states, node_states, sizeof(struct belief) * num_vertices, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->edges_messages, current_messages, sizeof(struct belief) * num_edges, hipMemcpyDeviceToHost));

    CUDA_CHECK_RETURN(hipFree(edges_joint_probabilities));
    CUDA_CHECK_RETURN(hipFree(current_messages));
    CUDA_CHECK_RETURN(hipFree(node_states));

    CUDA_CHECK_RETURN(hipFree(edges_src_index));
    CUDA_CHECK_RETURN(hipFree(edges_dest_index));

    CUDA_CHECK_RETURN(hipFree(dest_nodes_to_edges_nodes));
    CUDA_CHECK_RETURN(hipFree(dest_nodes_to_edges_edges));

    CUDA_CHECK_RETURN(hipFree(delta));
    CUDA_CHECK_RETURN(hipFree(delta_array));

    /*printf("After=====");
    print_nodes(graph);
    print_edges(graph);*/

    return num_iter;
}

/**
 * Runs loopy BP and outputs the result
 * @param expression The BNF expression holding the graph
 * @param file_name The file name of the graph data
 * @param out The file handle for the CSV file
 */
void run_test_loopy_belief_propagation_cuda(struct expression * expression, const char * file_name, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations;

    graph = build_graph(expression);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);

    start = clock();
    init_previous_edge(graph);

    num_iterations = loopy_propagate_until_cuda(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s,loopy,%d,%d,%d,%d,%lf\n", file_name, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}

/**
 * Runs loopy BP on the XML file
 * @param file_name The name of the XML file
 * @param out The file handle for the CSV file to output to
 */
void run_test_loopy_belief_propagation_xml_file_cuda(const char * file_name, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations;

    graph = parse_xml_file(file_name);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);

    start = clock();
    init_previous_edge(graph);

    num_iterations = loopy_propagate_until_cuda(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s,loopy,%d,%d,%d,%d,%lf\n", file_name, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}

/**
 * Runs the edge-optimized version of loopy BP
 * @param file_name The path of the file to read
 * @param out The file handle for the CSV output
 */
void run_test_loopy_belief_propagation_xml_file_edge_cuda(const char * file_name, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations;

    graph = parse_xml_file(file_name);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);

    start = clock();
    init_previous_edge(graph);

    num_iterations = loopy_propagate_until_cuda_edge(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s,loopy-edge,%d,%d,%d,%d,%lf\n", file_name, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}



/**
 * Runs loopy BP on the XML file
 * @param edge_file_name The file to read for the SNAP edges
 * @param node_file_name The file to read for the SNAP observed nodes
 * @param out The CSV file to output to
 */
void run_test_loopy_belief_propagation_snap_file_cuda(const char * edge_file_name, const char * node_file_name, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations;

    graph = parse_graph_from_snap_files(edge_file_name, node_file_name);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);

    start = clock();
    init_previous_edge(graph);

    num_iterations = loopy_propagate_until_cuda(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s-%s,loopy,%d,%d,%d,%d,%lf\n", edge_file_name, node_file_name, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}

/**
 * Runs the edge-optimized version of loopy BP
 * @param edge_file_name The file to read for the SNAP edges
 * @param node_file_name The file to read for the SNAP observed nodes
 * @param out The CSV file to output to
 */
void run_test_loopy_belief_propagation_snap_file_edge_cuda(const char * edge_file_name, const char * node_file_name, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations;

    graph = parse_graph_from_snap_files(edge_file_name, node_file_name);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);

    start = clock();
    init_previous_edge(graph);

    num_iterations = loopy_propagate_until_cuda_edge(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s-%s,loopy-edge,%d,%d,%d,%d,%lf\n", edge_file_name, node_file_name, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}


void run_test_loopy_belief_propagation_mtx_files_cuda(const char * edge_mtx, const char *node_mtx, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations;

    graph = build_graph_from_mtx(edge_mtx, node_mtx);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);

    start = clock();
    init_previous_edge(graph);

    num_iterations = loopy_propagate_until_cuda(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s-%s,loopy,%d,%d,%d,%d,%lf\n", edge_mtx, node_mtx, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}

void run_test_loopy_belief_propagation_mtx_files_cuda_partitioned(const char * edge_mtx, const char *node_mtx, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations, num_partitions;

    graph = build_graph_from_mtx(edge_mtx, node_mtx);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);
    //num_partitions = graph->current_num_vertices / PARTITION_SIZE;
    num_partitions = NUM_PARTITIONS;

    if(graph->current_num_vertices < num_partitions) {
        start = clock();
        init_previous_edge(graph);

        num_iterations = loopy_propagate_until_cuda(graph, PRECISION, NUM_ITERATIONS);
        end = clock();
    }
    else {
        partition_graph_by_nodes(graph, num_partitions);

        start = clock();
        init_previous_edge(graph);

        num_iterations = loopy_propagate_until_cuda_partitioned(graph, PRECISION, NUM_PARTITIONS, num_partitions);
        end = clock();
    }


    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s-%s,loopy-partitioned,%d,%d,%d,%d,%lf\n", edge_mtx, node_mtx, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}

void run_test_loopy_belief_propagation_mtx_files_edge_cuda(const char * edge_mtx, const char * node_mtx, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations;

    graph = build_graph_from_mtx(edge_mtx, node_mtx);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);

    start = clock();
    init_previous_edge(graph);

    num_iterations = loopy_propagate_until_cuda_edge(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s-%s,loopy-edge,%d,%d,%d,%d,%lf\n", edge_mtx, node_mtx, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}

/**
 * Checks that the CUDA kernel completed
 * @param file The source code file
 * @param line The line within the source code file that executes the kernel
 * @param statement The name of the kernel
 * @param err The error message
 */
void CheckCudaErrorAux (const char *file, unsigned int line, const char *statement, hipError_t err)
{
    if (err == hipSuccess) {
        return;
    }
    printf("%s returned %s (%d) at %s:%d\n", statement, hipGetErrorString(err), err, file, line);
    exit (1);
}

